
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cstdio>

#define CHECK_CUBLAS(Expr) { \
    int err = (Expr); \
    if (err != 0) { \
        printf("cuBLAS error %d at line %d\n", err, __LINE__); \
    } \
}

void gemm(hipblasHandle_t handle,
          int m,
          int n,
          int k,
          const void *alpha,
          const void *beta,
          hipDataType input_type,
          const void *A,
          const void *B,
          hipDataType output_type,
          void *C,
#if __CUDACC_VER_MAJOR__ >= 11
          cublasComputeType_t compute_type,
#else
          hipDataType compute_type,
#endif
          hipblasGemmAlgo_t algo) {
    CHECK_CUBLAS(hipblasGemmEx(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
        alpha, B, input_type, n, A, input_type, k,
        beta, C, output_type, n, compute_type, algo));
}

int main() {
    int m = 5120;
    int n = 4096;
    int k = 4096;

    float alpha = 1;
    float beta = 0;

    hipDataType input_type = HIP_R_32F;
    hipDataType output_type = HIP_R_32F;
#if __CUDACC_VER_MAJOR__ >= 11
    cublasComputeType_t compute_type = CUBLAS_COMPUTE_32F;
#else
    hipDataType compute_type = HIP_R_32F;
#endif
    hipblasGemmAlgo_t algo = HIPBLAS_GEMM_DEFAULT;

    int iter = 10;

    void *A, *B, *C;
    hipMalloc(&A, m * k * sizeof(float));
    hipMalloc(&B, k * n * sizeof(float));
    hipMalloc(&C, m * n * sizeof(float));

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // warmup
    gemm(handle, m, n, k, &alpha, &beta, input_type, A, B,
         output_type, C, compute_type, algo);

    hipEventRecord(start);
    for (int i = 0; i < iter; ++i) {
        gemm(handle, m, n, k, &alpha, &beta, input_type, A, B,
             output_type, C, compute_type, algo);
    }
    hipEventRecord(stop);

    float time_ms = 0.f;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_ms, start, stop);

    long ops = (long)m * n * k * 2;
    double gops = ((double)ops / 1e9) / ((double)time_ms / iter / 1e3);
    printf("%f Gops\n", gops);

    hipFree(A);
    hipFree(B);
    hipFree(C);
}



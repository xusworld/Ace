#include "hip/hip_runtime.h"
#include "saber/funcs/impl/cuda/saber_maxout.h"

namespace anakin {
namespace saber {

template <typename dtype>
__global__ void max_out(const dtype* input_ptr, dtype* output_ptr, const int count, 
                        const int num_out, const int c_out, const int h_out, const int w_out, const int groups) {
    
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int thread_num = blockDim.x * gridDim.x;
    int feature_size = h_out * w_out;
    int feature_map_size = feature_size * c_out;
    for (int i = tid; i < count; i += thread_num) {
        int batch_index = i / feature_map_size;
        int channel_index = (i / feature_size) % c_out;
        int feature_inner_index = i % feature_size;
        int src_index = (batch_index * feature_map_size + channel_index * feature_size) * groups + feature_inner_index;
        dtype max = input_ptr[src_index]; //get first element.
        for (int j = 1; j < groups; j++) {
            dtype tmp = input_ptr[src_index + j * feature_size];
            max = max < tmp ? tmp: max;
        }
        output_ptr[i] = max;
    }
}

template <DataType OpDtype>
SaberStatus SaberMaxOut<NV, OpDtype>::dispatch(const std::vector<Tensor<NV>*>& inputs,
    std::vector<Tensor<NV>*>& outputs,
    MaxOutParam<NV>& param) {
    hipStream_t hip_stream = this->_ctx->get_compute_stream();
    const OpDataType* input_ptr = (const OpDataType*)inputs[0]->data();
    OpDataType* output_ptr = (OpDataType*)outputs[0]->mutable_data();
    int count = outputs[0]->valid_size();
    max_out<OpDataType><<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, hip_stream>>>(
        input_ptr,
        output_ptr,
        count,
        _num_out,
        _c_out,
        _h_out,
        _w_out,
        param.groups    
    );

    CUDA_POST_KERNEL_CHECK;

    return SaberSuccess;
}

template class SaberMaxOut<NV, AK_FLOAT>;
DEFINE_OP_TEMPLATE(SaberMaxOut, MaxOutParam, NV, AK_HALF);
DEFINE_OP_TEMPLATE(SaberMaxOut, MaxOutParam, NV, AK_INT8);

} // namespace saber.
} // namespace anakin.
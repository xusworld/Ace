#include "hip/hip_runtime.h"
#include <limits>
#include "saber/funcs/impl/cuda/saber_softmax.h"

namespace anakin{

namespace saber{

//! general kernel for softmax
template <typename dtype>
__global__ void softmax_max_kernel(int total_size, const dtype* in_data, dtype* out_data, \
        dtype min_data, int inner_num, int outer_num, int axis_size){

    //! compute data index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_size) {
        int idx_inner = idx % inner_num;
        int idx_outer = (idx / inner_num) * axis_size;
        int real_index = idx_outer * inner_num + idx_inner;
        //! get maximum data across softmax axis
        dtype max_data = min_data;
        for (int i = 0; i < axis_size; ++i) {
            max_data = in_data[real_index] > max_data? in_data[real_index] : max_data;
            real_index += inner_num;
    }
        out_data[idx] = max_data;
    }
}

template <typename dtype>
__global__ void softmax_max_roi_kernel(int total_size, const dtype* in_data, \
        dtype* out_data, dtype min_data, \
        const int* input_stride_real, const int* output_stride_real, const int* shape_valid, \
        int softmax_axis, int axis_size, int dims){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_size) {

        //! compute real data index
        int input_real_index = 0;
        for (int i = dims - 1; i >= 0; i--) {
            if (i == softmax_axis) {
                continue;
            } else {
                int x = idx % shape_valid[i];
                input_real_index += x * input_stride_real[i];
                idx = idx / shape_valid[i];
            }
        }

        //! get maximum data across softmax axis
        dtype max_data = min_data;
        for (int i = 0; i < axis_size; ++i) {
            max_data = in_data[input_real_index] > max_data? \
                    in_data[input_real_index] : max_data;
            input_real_index += i * input_stride_real[softmax_axis];
        }
        out_data[idx] = max_data;
    }
}

template <typename dtype>
__global__ void softmax_sub_exp_sum_kernel(int total_size, const dtype* in_data, \
        dtype* out_data, const dtype* max_data, dtype* sum_data, \
        int inner_num, int outer_num, int axis_size){

    //! compute data index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < total_size) {
        int idx_inner = idx % inner_num;
        int idx_outer = (idx / inner_num) * axis_size;

        dtype max_data_cur = max_data[idx];
        //dtype *sum_data_cur = &sum_data[idx];
        dtype sum_data_cur = 0;
        int real_index = idx_outer * inner_num + idx_inner;
        //! compute exp and summarize across the softmax axis
        for (int i = 0; i < axis_size; ++i) {
            dtype sub_data = in_data[real_index] - max_data_cur;
            sub_data = expf(sub_data);
            sum_data_cur += sub_data;
            out_data[real_index] = sub_data;
            real_index += inner_num;
        }
        sum_data[idx] = sum_data_cur;
    }
}

template <typename dtype>
__global__ void softmax_sub_exp_sum_roi_kernel(int total_size, \
        const dtype* in_data, dtype* out_data, \
        const dtype* max_data, dtype* sum_data, \
        const int* input_stride_real, const int* output_stride_real, const int* shape_valid, \
        int softmax_axis, int axis_size, int dims){

    //! compute data index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_size) {
        //! compute real data index
        int output_real_index = 0;
        for (int i = dims - 1; i >= 0; i--) {
            if (i == softmax_axis) {
                continue;
            } else {
                int x = idx % shape_valid[i];
                output_real_index += x * output_stride_real[i];
                idx = idx / shape_valid[i];
            }
        }

        dtype max_data_cur = max_data[idx];
        //dtype *sum_data_cur = &sum_data[idx];
        dtype sum_data_cur = 0;
        //! compute exp and summarize across the softmax axis
        for (int i = 0; i < axis_size; ++i) {
            dtype sub_data = in_data[output_real_index] - max_data_cur;
            sub_data = expf(sub_data);
            sum_data_cur += sub_data;
            out_data[output_real_index] = sub_data;
            output_real_index += output_stride_real[softmax_axis];
        }
        sum_data[idx] = sum_data_cur;
    }
}

template <typename dtype>
__global__ void softmax_divid_output_kernel(int total_size, dtype* io_data, \
        const dtype* sum_data, int inner_num, int outer_num, int axis_size){
    //! compute data index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_size) {
        int idx_inner = idx % inner_num;
        int idx_outer = (idx / inner_num) * axis_size;
        dtype sum_data_cur = sum_data[idx];
        int real_index = idx_outer * inner_num + idx_inner;
        //! compute final result
        for (int i = 0; i < axis_size; ++i) {
            io_data[real_index] = io_data[real_index] / sum_data_cur;
            real_index += inner_num;
        }
    }
}

template <typename dtype>
__global__ void softmax_divid_output_roi_kernel(int total_size, \
        dtype* io_data, const dtype* sum_data, \
        const int* input_stride_real, const int* output_stride_real, const int* shape_valid, \
        int softmax_axis, int axis_size, int dims){
    //! compute data index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_size) {
        //! compute real data index
        int output_real_index = 0;
        for (int i = dims - 1; i >= 0; i--) {
            if (i == softmax_axis) {
                continue;
            } else {
                int x = idx % shape_valid[i];
                output_real_index += x * output_stride_real[i];
                idx = idx / shape_valid[i];
            }
        }

        dtype sum_data_cur = sum_data[idx];
        //! compute final result
        for (int i = 0; i < axis_size; ++i) {
            io_data[output_real_index] = io_data[output_real_index] / sum_data_cur;
            output_real_index += output_stride_real[softmax_axis];
        }
    }
}

extern __shared__ char tile[];
template <typename dtype>
__global__ void sharemem_softmax_kernel(int total_size, \
        const dtype* in_data, dtype* out_data, \
        int inner_num, int outer_num, int axis_size){

    //__shared__ dtype data[MAX_AXIS_SIZE][CUDA_NUM_THREADS];
    dtype* data = (dtype*)tile + threadIdx.x;

    //! compute thread index and real data index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < total_size) {
        int idx_inner = idx % inner_num;
        int idx_outer = (idx / inner_num) * axis_size;
        int blocksize = blockDim.x;

        int real_index = idx_outer * inner_num + idx_inner;
        int loop_idx = real_index;
        //! read all data to sharemem in softmax channel
        #pragma unroll
        for (int i = 0; i < axis_size; ++i) {
            data[i * blocksize] = in_data[loop_idx];
            loop_idx += inner_num;
        }

        //! get maximum value in softmax channel
        dtype max_data = data[0];
        #pragma unroll
        for (int i = 1; i < axis_size; ++i) {
            dtype dt = data[i * blocksize];
            if (max_data < dt){
                max_data = dt;
            }
        }

        //! subtract then summarize
        dtype sum = 0;
        #pragma unroll
        for (int i = 0; i < axis_size; ++i) {
            //dtype *dt = &data[i][thread_idx];
            dtype *dt = data + i * blocksize;
            *dt = expf(*dt - max_data);
            sum += *dt;
        }

        //! write back result
        loop_idx = real_index;
        #pragma unroll
        for (int i = 0; i < axis_size; ++i) {
            out_data[loop_idx] = data[i * blocksize] / sum;
            loop_idx += inner_num;
        }
    }
}

template <typename dtype>
__global__ void sharemem_softmax_roi_kernel(int total_size, \
        const dtype* in_data, dtype* out_data, \
        const int* input_stride_real, const int* output_stride_real, const int* shape_valid, \
        int softmax_axis, int axis_size, int dims){

    //__shared__ dtype data[MAX_AXIS_SIZE][CUDA_NUM_THREADS];
    dtype* data = (dtype*)tile + threadIdx.x;

    //! compute thread index and real data index
    int idx1 = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = idx1;

    if (idx < total_size) {

        int blocksize = blockDim.x;

        //! compute real data index
        int input_real_index = 0;
        int output_real_index = 0;
        for (int i = dims - 1; i >= 0; i--) {
            if (i == softmax_axis) {
                continue;
            } else {
                int x = idx % shape_valid[i];
                input_real_index += x * input_stride_real[i];
                output_real_index += x * output_stride_real[i];
                idx = idx / shape_valid[i];
            }
        }

        //! read all data to sharemem in softmax channel
        #pragma unroll
        for (int i = 0; i < axis_size; ++i) {
            data[i * blocksize] = in_data[input_real_index];
            input_real_index += input_stride_real[softmax_axis];
    }

        //! get maximum value in softmax channel
        dtype max_data = data[0];
        #pragma unroll
        for (int i = 1; i < axis_size; ++i) {
            dtype dt = data[i * blocksize];
            if (max_data < dt){
                max_data = dt;
            }
        }

        //! subtract then summarize
        dtype sum = 0;
        #pragma unroll
        for (int i = 0; i < axis_size; ++i) {
            //dtype *dt = &data[i][thread_idx];
            dtype *dt = data + i * blocksize;
            *dt = expf(*dt - max_data);
            sum += *dt;
        }

        //! write back result
        #pragma unroll
        for (int i = 0; i < axis_size; ++i) {
            out_data[output_real_index] = data[i * blocksize] / sum;
            output_real_index += output_stride_real[softmax_axis];
        }
    }
}

template <>
SaberStatus SaberSoftmax<NV, AK_FLOAT>::create(
        const std::vector<Tensor<NV> *>& inputs,
        std::vector<Tensor<NV> *>& outputs,
        SoftmaxParam<NV>& param, Context<NV>& ctx) {

    //! compute size
    Shape shape_in = inputs[0]->valid_shape();
    Shape shape_out = outputs[0]->valid_shape();
    CHECK_EQ(shape_in == shape_out, true) << "valid shapes must be the same";
    _outer_num = inputs[0]->count_valid(0, param.axis);
    _inner_num = inputs[0]->count_valid(param.axis + 1, inputs[0]->dims());
    _axis_size = shape_in[param.axis];

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, API::get_device_id());
    size_t sharedmem_size = deviceProp.sharedMemPerBlock;
    _max_dimsize = sharedmem_size / sizeof(float) / CUDA_NUM_THREADS;

    Shape sh_tmp({1, 1, 1, _outer_num * _inner_num});
    if (_axis_size > _max_dimsize){
        //! re_alloc device memory
        _max_data.reshape(sh_tmp);
        _sum_data.reshape(sh_tmp);
    }

    //! CHECK whether the input or output tensor is with continuous buffer or not
    _is_continue_buf = outputs[0]->is_continue_mem() && inputs[0]->is_continue_mem();
    _dims = shape_in.size();
    if (!_is_continue_buf) {
        Shape sh_input_real_stride = inputs[0]->get_stride();
        Shape sh_output_real_stride = outputs[0]->get_stride();

        //! re_alloc device memory
        Shape sh({1, 1, 1, _dims});
        _valid_shape.reshape(sh);
        _input_stride.reshape(sh);
        _output_stride.reshape(sh);

        CUDA_CHECK(hipMemcpy(_valid_shape.mutable_data(), inputs[0]->valid_shape().data(), \
                sizeof(int) * _dims, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(_input_stride.mutable_data(), sh_input_real_stride.data(), \
                sizeof(int) * _dims, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(_output_stride.mutable_data(), sh_output_real_stride.data(), \
                sizeof(int) * _dims, hipMemcpyHostToDevice));
    }
    return SaberSuccess;
}

template <>
SaberStatus SaberSoftmax<NV, AK_FLOAT>::init(
    const std::vector<Tensor<NV> *>& inputs,
    std::vector<Tensor<NV> *>& outputs,
    SoftmaxParam<NV>& param, Context<NV>& ctx) {

    //! get context
    this->_ctx = &ctx;
    return create(inputs, outputs, param, ctx);
}


template <>
SaberStatus SaberSoftmax<NV, AK_FLOAT>::dispatch(\
    const std::vector<Tensor<NV> *>& inputs, \
    std::vector<Tensor<NV> *>& outputs, \
    SoftmaxParam<NV>& param) {

    hipStream_t stream = this->_ctx->get_compute_stream();
    //! inputs only has one tensor
    int total_threads = this->_inner_num * this->_outer_num;
    const float* data_in = (const float* )inputs[0]->data();
    float* data_out = (float*)outputs[0]->mutable_data();
    float* max_data = (float*)this->_max_data.mutable_data();
    float* sum_data = (float*)this->_sum_data.mutable_data();
    const int* valid_shape = (const int*)_valid_shape.data();  
    const int* input_stride = (const int*)_input_stride.data();
    const int* output_stride = (const int*)_output_stride.data();

    if (_is_continue_buf) {
        //! softmax kernel without roi
        if (this->_axis_size <= _max_dimsize){
            int sharemem_size = this->_axis_size * CUDA_NUM_THREADS * sizeof(float);
            sharemem_softmax_kernel<float>\
                <<<CUDA_GET_BLOCKS(total_threads), CUDA_NUM_THREADS, sharemem_size, stream>>>(
                    total_threads, data_in, data_out,
                            this->_inner_num, this->_outer_num, this->_axis_size);
        } else {
            //! firstly, get maximum data
            float min_data = std::numeric_limits<float>::min();
            softmax_max_kernel<float>\
                <<<CUDA_GET_BLOCKS(total_threads), CUDA_NUM_THREADS, 0, stream>>>(
                    total_threads, data_in, max_data, min_data, \
                this->_inner_num, this->_outer_num, this->_axis_size);
            //! then, compute exp and sum data
            softmax_sub_exp_sum_kernel<float>
                    <<<CUDA_GET_BLOCKS(total_threads), CUDA_NUM_THREADS, 0, stream>>>(
                    total_threads, data_in, data_out, max_data, sum_data, \
                this->_inner_num, this->_outer_num, this->_axis_size);
            //! lastly, compute divided output
            softmax_divid_output_kernel<float>\
                <<<CUDA_GET_BLOCKS(total_threads), CUDA_NUM_THREADS, 0, stream>>>(
                    total_threads, data_out, sum_data, \
                this->_inner_num, this->_outer_num, this->_axis_size);
        }
    } else {
        //! softmax kernel with roi
        if (this->_axis_size <= _max_dimsize){
            int sharemem_size = this->_axis_size * CUDA_NUM_THREADS * sizeof(float);
            sharemem_softmax_roi_kernel<float>\
                <<<CUDA_GET_BLOCKS(total_threads), CUDA_NUM_THREADS, sharemem_size, stream>>>(
                    total_threads, data_in, data_out,
                    input_stride, output_stride, valid_shape, \
                    param.axis, _axis_size, _dims);
        } else {
            //! firstly, get maximum data
            float min_data = std::numeric_limits<float>::min();
            softmax_max_roi_kernel<float>\
                <<<CUDA_GET_BLOCKS(total_threads), CUDA_NUM_THREADS, 0, stream>>>(
                    total_threads, data_in, max_data, min_data, \
                    input_stride, output_stride, valid_shape, \
                    param.axis, _axis_size, _dims);
            //! then, compute exp and sum data
            softmax_sub_exp_sum_roi_kernel<float>
                    <<<CUDA_GET_BLOCKS(total_threads), CUDA_NUM_THREADS, 0, stream>>>(
                    total_threads, data_in, data_out, max_data, sum_data, \
                    input_stride, output_stride, valid_shape, \
                    param.axis, _axis_size, _dims);
            //! lastly, compute divided output
            softmax_divid_output_roi_kernel<float>\
                <<<CUDA_GET_BLOCKS(total_threads), CUDA_NUM_THREADS, 0, stream>>>(
                    total_threads, data_out, sum_data, \
                    input_stride, output_stride, valid_shape, \
                    param.axis, _axis_size, _dims);
        }
    }

    return SaberSuccess;
}

// ============================================= int8
template <>
SaberStatus SaberSoftmax<NV, AK_INT8>::create(
        const std::vector<Tensor<NV> *>& inputs,
        std::vector<Tensor<NV> *>& outputs,
        SoftmaxParam<NV>& param, Context<NV>& ctx) {

    return SaberSuccess;
}

template <>
SaberStatus SaberSoftmax<NV, AK_INT8>::init(
        const std::vector<Tensor<NV> *>& inputs,
        std::vector<Tensor<NV> *>& outputs,
        SoftmaxParam<NV>& param, Context<NV>& ctx) {

    this->_ctx = &ctx;
    return create(inputs, outputs, param, ctx);
}

template <>
SaberStatus SaberSoftmax<NV, AK_INT8>::dispatch(
        const std::vector<Tensor<NV> *>& inputs,
        std::vector<Tensor<NV> *>& outputs,
        SoftmaxParam<NV>& param) {

    return SaberSuccess;
}

template class SaberSoftmax<NV, AK_FLOAT>;
template class SaberSoftmax<NV, AK_INT8>;
DEFINE_OP_TEMPLATE(SaberSoftmax, SoftmaxParam, NV, AK_HALF);
} //namespace anakin

} //namespace anakin

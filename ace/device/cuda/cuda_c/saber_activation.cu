#include "hip/hip_runtime.h"
#include "saber/funcs/impl/cuda/saber_activation.h"
#include "saber/core/tensor_op.h"
#include "saber/funcs/calibrate.h"

#define BUILD_DEV __device__

namespace anakin{
namespace saber{

template<typename Dtype>
__global__ void ker_relu_fwd(Dtype * out_data,
                             const Dtype* in_data, const int count, Dtype neg_slop,
                             int in_n, int in_c, int in_h, int in_w,
                             int in_n_stride, int in_c_stride, int in_h_stride, int in_w_stride,
                             int out_n_stride, int out_c_stride, int out_h_stride, int out_w_stride) {
    CUDA_KERNEL_LOOP(tid, count) {
        int w =  tid % in_w;
        int h = (tid / (in_w)) % in_h;
        int c = (tid / (in_h * in_w)) % in_c;
        int n = (tid / (in_c * in_h * in_w)) % in_n;

        int in_idx = n * in_n_stride
                     + c * in_c_stride
                     + h * in_h_stride
                     + w * in_w_stride;

        int out_idx =  n * out_n_stride
                       + c * out_c_stride
                       + h * out_h_stride
                       + w * out_w_stride;

        Dtype in_var = in_data[in_idx];
        out_data[out_idx] = in_var > Dtype(0) ? in_var : in_var * neg_slop;
    }
}

template<typename Dtype>
__global__ void ker_sigmoid_fwd(Dtype * out_data,
                                const Dtype* in_data, const int count,
                                int in_n, int in_c, int in_h, int in_w,
                                int in_n_stride, int in_c_stride, int in_h_stride, int in_w_stride,
                                int out_n_stride, int out_c_stride, int out_h_stride, int out_w_stride) {

    CUDA_KERNEL_LOOP(tid, count) {
        int w =  tid % in_w;
        int h = (tid / (in_w)) % in_h;
        int c = (tid / (in_h * in_w)) % in_c;
        int n = (tid / (in_c * in_h * in_w)) % in_n;

        int in_idx =   n * in_n_stride
                       + c * in_c_stride
                       + h * in_h_stride
                       + w * in_w_stride;

        int out_idx =   n * out_n_stride
                        + c * out_c_stride
                        + h * out_h_stride
                        + w * out_w_stride;

        Dtype in_var = in_data[in_idx];

        out_data[out_idx] = Dtype( Dtype(1) / (Dtype(1)+ exp(-in_var)));

    }
}

template<typename Dtype>
__global__ void ker_tanh_fwd(Dtype * out_data,
                             const Dtype* in_data, const int count,
                             int in_n, int in_c, int in_h, int in_w,
                             int in_n_stride, int in_c_stride, int in_h_stride, int in_w_stride,
                             int out_n_stride, int out_c_stride, int out_h_stride, int out_w_stride) {

    CUDA_KERNEL_LOOP(tid, count) {
        int w =  tid % in_w;
        int h = (tid / (in_w)) % in_h;
        int c = (tid / (in_h * in_w)) % in_c;
        int n = (tid / (in_c * in_h * in_w)) % in_n;

        int in_idx =   n * in_n_stride
                       + c * in_c_stride
                       + h * in_h_stride
                       + w * in_w_stride;

        int out_idx =   n * out_n_stride
                        + c * out_c_stride
                        + h * out_h_stride
                        + w * out_w_stride;

        Dtype in_var = in_data[in_idx];
        //(expf(in_var) - expf(-in_var)) / (expf(in_var) + expf(-in_var));exp
        out_data[out_idx] = Dtype(1) - (Dtype(2) / (Dtype(1) + exp(in_var * 2))); 

    }
}

template<typename Dtype>
__global__ void ker_stanh_fwd(Dtype * out_data,
                             const Dtype* in_data, const int count, const Dtype slope, const Dtype coef, 
                             int in_n, int in_c, int in_h, int in_w,
                             int in_n_stride, int in_c_stride, int in_h_stride, int in_w_stride,
                             int out_n_stride, int out_c_stride, int out_h_stride, int out_w_stride) {

    CUDA_KERNEL_LOOP(tid, count) {
        int w =  tid % in_w;
        int h = (tid / (in_w)) % in_h;
        int c = (tid / (in_h * in_w)) % in_c;
        int n = (tid / (in_c * in_h * in_w)) % in_n;

        int in_idx =   n * in_n_stride
                       + c * in_c_stride
                       + h * in_h_stride
                       + w * in_w_stride;

        int out_idx =   n * out_n_stride
                        + c * out_c_stride
                        + h * out_h_stride
                        + w * out_w_stride;


        Dtype in_var = in_data[in_idx];
        Dtype var = in_var * slope;
        //output_data[j] = param.coef * tanh(param.negative_slope * input_data[j]);
        out_data[out_idx] = Dtype( coef * (Dtype(1) - (Dtype(2) / (Dtype(1) + exp(var * 2)))));
    }
}

template<typename Dtype>
__global__ void ker_clipped_relu_fwd(Dtype * out_data,
                                     const Dtype* in_data, const int count, Dtype clipped_threadhold,
                                     int in_n, int in_c, int in_h, int in_w,
                                     int in_n_stride, int in_c_stride, int in_h_stride, int in_w_stride,
                                     int out_n_stride, int out_c_stride, int out_h_stride, int out_w_stride) {
    CUDA_KERNEL_LOOP(tid, count) {
        int w =  tid % in_w;
        int h = (tid / (in_w)) % in_h;
        int c = (tid / (in_h * in_w)) % in_c;
        int n = (tid / (in_c * in_h * in_w)) % in_n;

        int in_idx =   n * in_n_stride
                       + c * in_c_stride
                       + h * in_h_stride
                       + w * in_w_stride;

        int out_idx =   n * out_n_stride
                        + c * out_c_stride
                        + h * out_h_stride
                        + w * out_w_stride;

        Dtype in_var = in_data[in_idx];
        in_var = in_var > 0 ? in_var : 0;
        out_data[out_idx] = in_var < clipped_threadhold? in_var : clipped_threadhold;
    }
}

template<typename Dtype>
__global__ void ker_swish_fwd(Dtype * out_data,
                                     const Dtype* in_data, const int count, Dtype beta,
                                     int in_n, int in_c, int in_h, int in_w,
                                     int in_n_stride, int in_c_stride, int in_h_stride, int in_w_stride,
                                     int out_n_stride, int out_c_stride, int out_h_stride, int out_w_stride) {
    CUDA_KERNEL_LOOP(tid, count) {
        int w =  tid % in_w;
        int h = (tid / (in_w)) % in_h;
        int c = (tid / (in_h * in_w)) % in_c;
        int n = (tid / (in_c * in_h * in_w)) % in_n;

        int in_idx =   n * in_n_stride
                       + c * in_c_stride
                       + h * in_h_stride
                       + w * in_w_stride;

        int out_idx =   n * out_n_stride
                        + c * out_c_stride
                        + h * out_h_stride
                        + w * out_w_stride;

        Dtype in_var = in_data[in_idx];
        out_data[out_idx] = Dtype( in_var / (Dtype(1)+ exp(-(beta * in_var))));
    }
}

template<typename Dtype>
__global__ void ker_elu_fwd(Dtype * out_data,
                            const Dtype* in_data, const int count, Dtype coef,
                            int in_n, int in_c, int in_h, int in_w,
                            int in_n_stride, int in_c_stride, int in_h_stride, int in_w_stride,
                            int out_n_stride, int out_c_stride, int out_h_stride, int out_w_stride) {
    CUDA_KERNEL_LOOP(tid, count){
        int w =  tid % in_w;
        int h = (tid / (in_w)) % in_h;
        int c = (tid / (in_h * in_w)) % in_c;
        int n = (tid / (in_c * in_h * in_w)) % in_n;

        int in_idx =   n * in_n_stride
                       + c * in_c_stride
                       + h * in_h_stride
                       + w * in_w_stride;

        int out_idx =   n * out_n_stride
                        + c * out_c_stride
                        + h * out_h_stride
                        + w * out_w_stride;

        Dtype in_var = in_data[in_idx];
        out_data[out_idx] = in_var > 0 ? in_var : coef * (exp(in_var)-1);
    }
}

template<typename Dtype>
__global__ void ker_gelu_fwd(Dtype * out_data,
                            const Dtype* in_data, const int count,
                            int in_n, int in_c, int in_h, int in_w,
                            int in_n_stride, int in_c_stride, int in_h_stride, int in_w_stride,
                            int out_n_stride, int out_c_stride, int out_h_stride, int out_w_stride) {
    CUDA_KERNEL_LOOP(tid, count){
        int w =  tid % in_w;
        int h = (tid / (in_w)) % in_h;
        int c = (tid / (in_h * in_w)) % in_c;
        int n = (tid / (in_c * in_h * in_w)) % in_n;

        int in_idx =   n * in_n_stride
                       + c * in_c_stride
                       + h * in_h_stride
                       + w * in_w_stride;

        int out_idx =   n * out_n_stride
                        + c * out_c_stride
                        + h * out_h_stride
                        + w * out_w_stride;

        Dtype in_var = in_data[in_idx];
        Dtype coeff = 0.5 * (std::erf(in_var / pow(2, 0.5)) + 1);
        out_data[out_idx] = in_var  * coeff;
    }
}

template<typename Dtype>
__global__ void ker_prelu_fwd(Dtype * out_data,
                              const Dtype* in_data, const int count,
                              const Dtype* slope, bool is_channel_shared,
                              int in_n, int in_c, int in_h, int in_w,
                              int in_n_stride, int in_c_stride, int in_h_stride, int in_w_stride,
                              int out_n_stride, int out_c_stride, int out_h_stride, int out_w_stride) {
    CUDA_KERNEL_LOOP(tid, count){
        int w =  tid % in_w;
        int h = (tid / (in_w)) % in_h;
        int c = (tid / (in_h * in_w)) % in_c;
        int n = (tid / (in_c * in_h * in_w)) % in_n;

        int in_idx =   n * in_n_stride
                       + c * in_c_stride
                       + h * in_h_stride
                       + w * in_w_stride;

        int out_idx =   n * out_n_stride
                        + c * out_c_stride
                        + h * out_h_stride
                        + w * out_w_stride;

        Dtype in_var = in_data[in_idx];
        if (is_channel_shared) {
            out_data[out_idx] = in_var > 0 ? in_var : slope[0] * in_var;
        } else {
            out_data[out_idx] = in_var > 0 ? in_var : slope[c] * in_var;
        }
    }
}

template <>
SaberStatus SaberActivation<NV, AK_FLOAT>::create( \
        const std::vector<Tensor<NV>*>& inputs,
        std::vector<Tensor<NV>*>& outputs,
        ActivationParam<NV>& param, Context<NV>& ctx) {

    this->_ctx = &ctx;
    return SaberSuccess;
}

template <>
SaberStatus SaberActivation<NV, AK_FLOAT>::init( \
        const std::vector<Tensor<NV>*>& inputs,
        std::vector<Tensor<NV>*>& outputs,
        ActivationParam<NV>& param, Context<NV>& ctx) {

    this->_ctx = &ctx;
    return create(inputs, outputs, param, ctx);
}

template <>
SaberStatus SaberActivation<NV, AK_FLOAT>::dispatch( \
        const std::vector<Tensor<NV>*>& inputs,
        std::vector<Tensor<NV>*>& outputs,
        ActivationParam<NV>& param) {
    Shape in_shape = inputs[0]->valid_shape();
    Shape out_shape = outputs[0]->valid_shape();

    Shape stride_in = inputs[0]->get_stride();
    Shape stride_out = outputs[0]->get_stride();

    const float *in_data = (const float*)inputs[0]->data();
    float *out_data = (float*)outputs[0]->mutable_data();

    const int count = inputs[0]->valid_size();
    hipStream_t hip_stream = this->_ctx->get_compute_stream();

    float negative_slope = param.negative_slope;
    float coef = param.coef;
    switch (param.active) {
        //x > 0 ? x : 0
        case Active_relu:

            ker_relu_fwd<float>
                    <<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, hip_stream>>>(
                    out_data, in_data, count, negative_slope,
                            in_shape[0], in_shape[1], in_shape[2], in_shape[3],
                            stride_in[0], stride_in[1], stride_in[2], stride_in[3],
                            stride_out[0], stride_out[1], stride_out[2], stride_out[3]);
            break;

        // sigmoid: 1/(exp(-x) + 1)
        case Active_sigmoid:

            ker_sigmoid_fwd<float>
                    <<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, hip_stream>>>(
                    out_data, in_data, count,
                            in_shape[0], in_shape[1], in_shape[2], in_shape[3],
                            stride_in[0], stride_in[1], stride_in[2], stride_in[3],
                            stride_out[0], stride_out[1], stride_out[2], stride_out[3]);
            break;

        // swish: x / (exp(-b * x) + 1)
        case Active_swish:

            ker_swish_fwd<float>
                    <<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, hip_stream>>>(
                    out_data, in_data, count, coef,
                            in_shape[0], in_shape[1], in_shape[2], in_shape[3],
                            stride_in[0], stride_in[1], stride_in[2], stride_in[3],
                            stride_out[0], stride_out[1], stride_out[2], stride_out[3]);
            break;

        // tanh : (exp(x) - exp(-x)) / (exp(x) + exp(-x))
        case Active_tanh:
        
            ker_tanh_fwd<float>
                    <<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, hip_stream>>>(
                    out_data, in_data, count,
                            in_shape[0], in_shape[1], in_shape[2], in_shape[3],
                            stride_in[0], stride_in[1], stride_in[2], stride_in[3],
                            stride_out[0], stride_out[1], stride_out[2], stride_out[3]);
            break;
        
        // stanh : b * \frac{e^{a * x} - e^{-a * x}}{e^{a * x} + e^{-a * x}}
        case Active_stanh:

            ker_stanh_fwd<float>
                    <<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, hip_stream>>>(
                    out_data, in_data, count, negative_slope, coef, 
                            in_shape[0], in_shape[1], in_shape[2], in_shape[3],
                            stride_in[0], stride_in[1], stride_in[2], stride_in[3],
                            stride_out[0], stride_out[1], stride_out[2], stride_out[3]);
            break;

        // x > 0 ? x : 0;
        // x < threshold ? x : threshold
        case Active_clipped_relu:

            ker_clipped_relu_fwd<float>
                    <<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, hip_stream>>>(
                    out_data, in_data, count, coef,
                            in_shape[0], in_shape[1], in_shape[2], in_shape[3],
                            stride_in[0], stride_in[1], stride_in[2], stride_in[3],
                            stride_out[0], stride_out[1], stride_out[2], stride_out[3]);
            break;

        //elu:  x > 0 ? x : coef * (exp(x) - 1)
        case Active_elu:

            ker_elu_fwd<float>
                    <<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, hip_stream>>>(
                    out_data, in_data, count, coef,
                            in_shape[0], in_shape[1], in_shape[2], in_shape[3],
                            stride_in[0], stride_in[1], stride_in[2], stride_in[3],
                            stride_out[0], stride_out[1], stride_out[2], stride_out[3]);
            break;
        //gelu: x * 0.5(erf(x/sqrt(2)) + 1)
        case Active_gelu:
            ker_gelu_fwd<float>
                    <<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, hip_stream>>>(
                            out_data, in_data, count, 
                            in_shape[0], in_shape[1], in_shape[2], in_shape[3],
                            stride_in[0], stride_in[1], stride_in[2], stride_in[3],
                            stride_out[0], stride_out[1], stride_out[2], stride_out[3]);
            break;

        //prelu: x > 0 ? x : slope[c] * x
        case Active_prelu:
            auto prelu_param  = param.prelu_param;
            const float* slope_ptr = (const float*)prelu_param.slope->data();
            bool shared = prelu_param.channel_shared;
            ker_prelu_fwd<float>
                    <<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, hip_stream>>>(
                            out_data, in_data, count, 
                            slope_ptr, shared,
                            in_shape[0], in_shape[1], in_shape[2], in_shape[3],
                            stride_in[0], stride_in[1], stride_in[2], stride_in[3],
                            stride_out[0], stride_out[1], stride_out[2], stride_out[3]);
            break;
    }
    CUDA_POST_KERNEL_CHECK;
    outputs[0]->set_seq_offset(inputs[0]->get_seq_offset());
    return SaberSuccess;
}

// =================================int8 ==================
class ReluDev{
public:
    static __device__ float run(float in, float negative_slope, float placeholder) {
        return (in > 0.f) ? in : in * negative_slope;
    }
};
class SigmoidDev{
public:
    static __device__ float run(float in, float placeholder1, float placeholder2) {
        return float( float(1) / (float(1)+ exp(-in)));
    }
};

template <typename Op>
__global__
void ker_act_fwd_fp32_to_int8(char* out_data, const float* in_data,
        int in_num, int in_channel_4, int in_height, int in_width,
        int in_n_stride, int in_c_stride, int in_h_stride, int in_w_stride,
        int out_n_stride, int out_c_stride, int out_h_stride, int out_w_stride,
        const float negtive_slope, const float coef, float scale, int count) {

    int load0, load1, load2, load3;
    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    int write_w = (gid) % in_width;
    int write_h = (gid / (out_h_stride)) % in_height;
    int write_c = (gid / (out_c_stride)) % in_channel_4;
    int write_n = (gid / (out_n_stride)) % in_num;

    int in_offset = write_n * in_n_stride
                    + write_c * in_c_stride * 4
                    + write_h * in_h_stride
                    + write_w * in_w_stride;

    int out_offset = write_n * out_n_stride
                     + write_c * out_c_stride
                     + write_h * out_h_stride
                     + write_w;

    if (gid < count) {
        char4 write;
        float temp;
        temp = in_data[in_offset] * scale;
        temp = Op::run(temp, negtive_slope, coef);
        load0 = __float2int_rn(temp);
        write.x = static_cast<char>(load0);

        in_offset += in_c_stride;
        temp = in_data[in_offset] * scale;
        temp = Op::run(temp, negtive_slope, coef);
        load1 = __float2int_rn(temp);
        write.y = static_cast<char>(load1);

        in_offset += in_c_stride;
        temp = in_data[in_offset] * scale;
        temp = Op::run(temp, negtive_slope, coef);
        load2 = __float2int_rn(temp);
        write.z = static_cast<char>(load2);

        in_offset += in_c_stride;
        temp = in_data[in_offset] * scale;
        temp = Op::run(temp, negtive_slope, coef);
        load3 = __float2int_rn(temp);
        write.w = static_cast<char>(load3);

        ((char4*)out_data)[out_offset] = write;
    }
}

template <typename Op>
__global__
void ker_act_fwd_int8_to_fp32(float* out_data, const char* in_data,
        int in_num, int in_channel_4, int in_height, int in_width,
        int in_n_stride, int in_c_stride, int in_h_stride, int in_w_stride,
        int out_n_stride, int out_c_stride, int out_h_stride, int out_w_stride,
        const float negtive_slope, const float coef, const float scale, int count) {

    float load0, load1, load2, load3;
    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    int read_w = (gid) % in_width;
    int read_h = (gid / (in_h_stride)) % in_height;
    int read_c = (gid / (in_c_stride)) % in_channel_4;
    int read_n = (gid / (in_n_stride)) % in_num;

    int in_offset = read_n * in_n_stride
                    + read_c * in_c_stride
                    + read_h * in_h_stride
                    + read_w;

    int out_offset = read_n * out_n_stride
                     + read_c * (out_c_stride << 2)
                     + read_h * out_h_stride
                     + read_w * out_w_stride;

    if (gid < count) {
        char4 readin = ((const char4*)in_data)[in_offset];
        load0 = static_cast<float>(readin.x) * scale;
        load1 = static_cast<float>(readin.y) * scale;
        load2 = static_cast<float>(readin.z) * scale;
        load3 = static_cast<float>(readin.w) * scale;
        load0 = Op::run(load0, negtive_slope, coef);
        load1 = Op::run(load1, negtive_slope, coef);
        load2 = Op::run(load2, negtive_slope, coef);
        load3 = Op::run(load3, negtive_slope, coef);
        out_data[out_offset] = load0; out_offset += out_c_stride;
        out_data[out_offset] = load1; out_offset += out_c_stride;
        out_data[out_offset] = load2; out_offset += out_c_stride;
        out_data[out_offset] = load3;
    }
}

__global__ void ker_sigmoid_fwd_int8(char * out_data,
                                const char* in_data, const int count,
                                int in_n, int in_c, int in_h, int in_w,
                                int in_n_stride, int in_c_stride, int in_h_stride, int in_w_stride,
                                int out_n_stride, int out_c_stride, int out_h_stride, int out_w_stride,
                                float in_scale = 1.f, float out_scale = 1.f) {

    CUDA_KERNEL_LOOP(tid, count) {
        int w =  tid % in_w;
        int h = (tid / (in_w)) % in_h;
        int c = (tid / (in_h * in_w)) % in_c;
        int n = (tid / (in_c * in_h * in_w)) % in_n;

        int in_idx =   n * in_n_stride
                       + c * in_c_stride
                       + h * in_h_stride
                       + w * in_w_stride;

        int out_idx =   n * out_n_stride
                        + c * out_c_stride
                        + h * out_h_stride
                        + w * out_w_stride;

        char in_var = in_data[in_idx];
        float in = static_cast<float>(in_var) * in_scale;
        in = float( float(1) / (float(1)+ exp(-in)));
        in /= out_scale;
        out_data[out_idx] = static_cast<char>(in);
    }
}

template <>
SaberStatus SaberActivation<NV, AK_INT8>::create(
        const std::vector<Tensor<NV>*>& inputs,
        std::vector<Tensor<NV>*>& outputs,
        ActivationParam<NV>& param, Context<NV>& ctx) {

    this->_ctx = &ctx;
    if (inputs[0]->get_dtype() == AK_FLOAT) {
        Shape in_shape = inputs[0]->valid_shape();
        _int8_input.reshape(in_shape);
        _int8_input.set_scale(inputs[0]->get_scale());
        _int8_input.set_layout(Layout_NCHW_C4);
    }
    return SaberSuccess;
}

template <>
SaberStatus SaberActivation<NV, AK_INT8>::init(
        const std::vector<Tensor<NV>*>& inputs,
        std::vector<Tensor<NV>*>& outputs,
        ActivationParam<NV>& param, Context<NV>& ctx) {

    this->_ctx = &ctx;
    return create(inputs, outputs, param, ctx);
}

__global__ void ker_clipped_relu_fwd_s8s8(char * out_data,
                                  const char* in_data, const int count, float clipped_threadhold,
                                  int in_n, int in_c, int in_h, int in_w,
                                  int in_n_stride, int in_c_stride, int in_h_stride, int in_w_stride,
                                  int out_n_stride, int out_c_stride, int out_h_stride, int out_w_stride,
                                  float in_scale, float out_scale) {

    CUDA_KERNEL_LOOP(tid, count) {
        int w =  tid % in_w;
        int h = (tid / (in_w)) % in_h;
        int c = (tid / (in_h * in_w)) % in_c;
        int n = (tid / (in_c * in_h * in_w)) % in_n;

        int in_idx = n * in_n_stride
                     + c * in_c_stride
                     + h * in_h_stride
                     + w * in_w_stride;

        int out_idx =  n * out_n_stride
                       + c * out_c_stride
                       + h * out_h_stride
                       + w * out_w_stride;

        char in_var = in_data[in_idx];
        if (in_var < 0) {
            out_data[out_idx] = 0;
        } else {
            float temp = static_cast<float>(in_var) * in_scale;
            if (temp > clipped_threadhold) {
                temp = clipped_threadhold * in_scale / out_scale;
                out_data[out_idx] = static_cast<char>(__float2int_rn(temp));
            } else {
                out_data[out_idx] = in_var;
            }
        }
    }
}

__global__
void ker_clipped_relu_fwd_s8s8(void* out_data, const void* in_data, const float clipped_threadhold,
                         int valid_num, int valid_channel_4, int valid_height, int valid_width,
                         int in_n_stride, int in_c_stride, int in_h_stride, int in_w_stride,
                         int out_n_stride, int out_c_stride, int out_h_stride, int out_w_stride,
                         const float scale, const float out_scale, int count) {

    float load0, load1, load2, load3;
    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    int read_w = (gid) % valid_width;
    int read_h = (gid / (in_h_stride)) % valid_height;
    int read_c = (gid / (in_c_stride)) % valid_channel_4;
    int read_n = (gid / (in_n_stride)) % valid_num;

    int in_offset = read_n * in_n_stride
                    + read_c * in_c_stride
                    + read_h * in_h_stride
                    + read_w;

    if (gid < count) {

        char4 readin = __ldg(&((const char4*)in_data)[in_offset]);

        load0 = static_cast<float>(readin.x) * scale;
        load1 = static_cast<float>(readin.y) * scale;
        load2 = static_cast<float>(readin.z) * scale;
        load3 = static_cast<float>(readin.w) * scale;

        load0 = load0 > 0 ? load0 : 0;
        load0 = load0 < clipped_threadhold? load0 : clipped_threadhold;
        load1 = load1 > 0 ? load1 : 0;
        load1 = load1 < clipped_threadhold? load1 : clipped_threadhold;
        load2 = load2 > 0 ? load2 : 0;
        load2 = load2 < clipped_threadhold? load2 : clipped_threadhold;
        load3 = load3 > 0 ? load3 : 0;
        load3 = load3 < clipped_threadhold? load3 : clipped_threadhold;
        char4 store;

        store.x = static_cast<char>(__float2int_rn(load0 * out_scale));
        store.y = static_cast<char>(__float2int_rn(load1 * out_scale));
        store.z = static_cast<char>(__float2int_rn(load2 * out_scale));
        store.w = static_cast<char>(__float2int_rn(load3 * out_scale));

        ((char4*)out_data)[in_offset] = store;
    }
}

__global__
void ker_clipped_relu_fwd_s8f32(void* out_data, const void* in_data,
        const float clipped_threadhold,
        int valid_num, int valid_channel_4, int valid_height, int valid_width,
        int in_n_stride, int in_c_stride, int in_h_stride, int in_w_stride,
        int out_n_stride, int out_c_stride, int out_h_stride, int out_w_stride,
        const float scale, const float out_scale, int count) {

    float load0, load1, load2, load3;
    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    int read_w = (gid) % valid_width;
    int read_h = (gid / (in_h_stride)) % valid_height;
    int read_c = (gid / (in_c_stride)) % valid_channel_4;
    int read_n = (gid / (in_n_stride)) % valid_num;
    int scale_index = read_c << 2;

    int in_offset = read_n * in_n_stride
                    + read_c * in_c_stride
                    + read_h * in_h_stride
                    + read_w;

    int out_offset = read_n * out_n_stride
                     + read_c * (out_c_stride << 2)
                     + read_h * out_h_stride
                     + read_w * out_w_stride;

    if (gid < count) {

        char4 readin = __ldg(&((const char4*)in_data)[in_offset]);

        load0 = static_cast<float>(readin.x) * scale;
        load1 = static_cast<float>(readin.y) * scale;
        load2 = static_cast<float>(readin.z) * scale;
        load3 = static_cast<float>(readin.w) * scale;
        load0 = load0 > 0 ? load0 : 0;
        load0 = load0 < clipped_threadhold? load0 : clipped_threadhold;
        load1 = load1 > 0 ? load1 : 0;
        load1 = load1 < clipped_threadhold? load1 : clipped_threadhold;
        load2 = load2 > 0 ? load2 : 0;
        load2 = load2 < clipped_threadhold? load2 : clipped_threadhold;
        load3 = load3 > 0 ? load3 : 0;
        load3 = load3 < clipped_threadhold? load3 : clipped_threadhold;
        ((float*)out_data)[out_offset] = load0; out_offset += out_c_stride;
        ((float*)out_data)[out_offset] = load1; out_offset += out_c_stride;
        ((float*)out_data)[out_offset] = load2; out_offset += out_c_stride;
        ((float*)out_data)[out_offset] = load3;
    }
}

template <>
SaberStatus SaberActivation<NV, AK_INT8>::dispatch(
        const std::vector<Tensor<NV>*>& inputs,
        std::vector<Tensor<NV>*>& outputs,
        ActivationParam<NV>& param) {

    const void *in_data = inputs[0]->data();
    void *out_data = outputs[0]->mutable_data();

    const int count = inputs[0]->valid_size();
    int in_c_4 = inputs[0]->channel() / 4;
    int out_c_4 = outputs[0]->channel() / 4;

//    float negative_slope = param.negative_slope;
    float coef = param.coef;

    float in_scale = inputs[0]->get_scale()[0];
    float out_scale = 1.f / outputs[0]->get_scale()[0];

    Shape out_stride = outputs[0]->get_stride();
    Shape in_shape = inputs[0]->valid_shape();
    Shape out_shape = outputs[0]->valid_shape();
//    int count = in_shape[0] * in_shape[1] * in_shape[2] * in_shape[3];

    hipStream_t hip_stream = _ctx->get_compute_stream();

    if (inputs[0]->get_dtype() == AK_FLOAT) {
        conv_calibrate_fp32_int8_c4(_int8_input, *inputs[0], in_scale, *(this->_ctx));
        in_data = _int8_input.data();
    } else {
        in_data = inputs[0]->data();
    }

    if (outputs[0]->get_dtype() == AK_INT8) {
        switch (param.active) {
        case Active_clipped_relu:
            ker_clipped_relu_fwd_s8s8
                    <<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, hip_stream>>>(
                            out_data, in_data, coef,
                            in_shape[0], in_shape[1], in_shape[2], in_shape[3],
                            in_shape[1] * in_shape[2] * in_shape[3],
                            in_shape[2] * in_shape[3],
                            in_shape[3], 1,
                            out_stride[0], out_stride[1], out_stride[2], out_stride[3],
                            in_scale, out_scale, count);
            break;
        default:
            LOG(FATAL) << "Not implement this activation in this data config" << param.active;
            break;
        }
    } else if (outputs[0]->get_dtype() == AK_FLOAT) {
        switch (param.active) {
            case Active_clipped_relu:
                ker_clipped_relu_fwd_s8f32
                        <<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, hip_stream>>>(
                        out_data, in_data, coef,
                                in_shape[0], in_shape[1], in_shape[2], in_shape[3],
                                in_shape[1] * in_shape[2] * in_shape[3],
                                in_shape[2] * in_shape[3],
                                in_shape[3], 1,
                                out_stride[0], out_stride[1], out_stride[2], out_stride[3],
                                in_scale, out_scale, count);
                break;
            default:
                        LOG(FATAL) << "Not implement this activation in this data config" << param.active;
                break;
        }
    } else {
        LOG(FATAL) << "not supported yet!!!";
    }

    CUDA_POST_KERNEL_CHECK;
    return SaberSuccess;
}

template class SaberActivation<NV, AK_FLOAT>;
template class SaberActivation<NV, AK_INT8>;
DEFINE_OP_TEMPLATE(SaberActivation, ActivationParam, NV, AK_HALF);
}
}
